#include <stddef.h>
#include <iostream>
#include <hip/hip_runtime.h>

__global__
void vecAddKernel(float* A, float* B, float* C, size_t n) {
  // const auto i = threadIdx.x;
}

void vecAdd(float* A_h, float* B_h, float* C_h, size_t n) {

  const auto size = n * sizeof(float);
  std::cout << "Allocating " << size << " bytes" << std::endl;

  float* A_d;
  float* B_d;
  float* C_d;
  hipMalloc(&A_d, size);
  hipMalloc(&B_d, size);
  hipMalloc(&C_d, size);

  const hipError_t err_A = hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  const hipError_t err_B = hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
  std::cout << hipGetErrorString(err_A) << std::endl;
  std::cout << hipGetErrorString(err_B) << std::endl;

  for (size_t i = 0; i < n; ++i) {
    C_h[i] = A_h[i] + B_h[i];
  }

  const hipError_t err_C = hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
  std::cout << hipGetErrorString(err_C) << std::endl;

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}

int main() {
  size_t n = 5;
  float A[n];
  float B[n];
  float C[n];
  A[0] = 1.0;
  A[1] = 3.0;
  A[2] = 5.0;
  A[3] = 7.0;
  A[4] = 9.0;
  B[0] = 10.0;
  B[1] = 30.0;
  B[2] = 50.0;
  B[3] = 70.0;
  B[4] = 90.0;
  vecAdd(A, B, C, n);
  for (size_t i = 0; i < n; ++i) {
    std::cout << i << " " << C[i] << std::endl;
  }
  return 0;
}
