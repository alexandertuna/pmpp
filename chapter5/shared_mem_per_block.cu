
#include <hip/hip_runtime.h>
#include <iostream>

int main() {

  int device_count{0};
  hipGetDeviceCount(&device_count);
  printf("Device count: %i\n", device_count);

  for (int dev{0}; dev < device_count; ++dev) {
    hipDeviceProp_t prop;
    const auto err = hipGetDeviceProperties(&prop, dev);
    printf("Device %i: %s has %i sharedMemPerBlock\n", dev, prop.name, prop.sharedMemPerBlock);
  }
  
  printf("Getting device props ...\n");
  return 0;
}
