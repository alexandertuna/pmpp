
#include <hip/hip_runtime.h>
#include <stddef.h>
#include <iostream>

__global__
void vecAddKernel(float* A, float* B, float* C, size_t n) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    printf("i vecAddKernel %lu\n", i);
    C[i] = A[i] + B[i];
  }
}

void vecAdd(float* A_h, float* B_h, float* C_h, size_t n) {

  const auto size = n * sizeof(float);
  std::cout << "Allocating " << size << " bytes" << std::endl;

  float* A_d;
  float* B_d;
  float* C_d;
  hipMalloc(&A_d, size);
  hipMalloc(&B_d, size);
  hipMalloc(&C_d, size);

  const hipError_t err_A = hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  const hipError_t err_B = hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
  std::cout << hipGetErrorString(err_A) << std::endl;
  std::cout << hipGetErrorString(err_B) << std::endl;

  std::cout << "Adding on GPU" << std::endl;
  const dim3 n_blocks(4, 1, 1);
  const dim3 n_threads(256, 1, 1);
  vecAddKernel<<<n_blocks, n_threads>>>(A_d, B_d, C_d, n);

  const hipError_t err_C = hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
  std::cout << hipGetErrorString(err_C) << std::endl;

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}

int main() {
  size_t n = 5;
  float A[n];
  float B[n];
  float C[n];
  A[0] = 1.0;
  A[1] = 3.0;
  A[2] = 5.0;
  A[3] = 7.0;
  A[4] = 9.0;
  B[0] = 10.0;
  B[1] = 30.0;
  B[2] = 50.0;
  B[3] = 70.0;
  B[4] = 90.0;
  vecAdd(A, B, C, n);
  for (size_t i = 0; i < n; ++i) {
    std::cout << i << " " << C[i] << std::endl;
  }
  return 0;
}
